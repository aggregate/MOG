#include "hip/hip_runtime.h"
/*	mog.cu

	MIMD On GPU simulator by Hank Dietz

	This simulator uses an accumulator-based instruction set that
	is based on reprocessing MIPSEL assembly code.  The instruction
	set is fundamentally similar, but has been simplified to make
	decoding instructions trivial.

	20101122 first alpha test release
*/

#define	VERSION	20101122
#define	TARGET	"cuda"

#define	TRACE
#undef	TRACE

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>

//#include <cutil.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

#include "op.h"
#include "do.h"
#include "pr.h"

#include "mog.h"

#define	GPU_MULTIPROCESSORS	4
#define	GPU_WARPSIZE		32
#define	GPU_THREADSPERBLOCK	(8*GPU_WARPSIZE)

#define	SHARED_SKEW	16

#define	WARPSIZE	32		/* Size of a WARP on this GPU */
#define	WARPDIV(X)	((X)>>5)	/* X / WARPSIZE */
#define	WARPMOD(X)	((X)&31)	/* X % WARPSIZE */
#define	WARPMUL(X)	((X)<<5)	/* X * WARPSIZE */

#define	MAR(X)		(((X)&~3)<<3)	/* X * WARPSIZE / 4 */

/*	TIMEOUT can be anything; 1 for TRACE */
#ifdef	TRACE
#define	TIMEOUT		1
#else
#define	TIMEOUT		(1024)
#endif

#define	BNPROC		GPU_THREADSPERBLOCK
#define	NPROC		(GPU_MULTIPROCESSORS*BNPROC)

typedef union word_union {
	float			f;
	int			i;
	unsigned int		u;
	short			h[2];
	signed char		b[4];
} word_t;

#define	FLAGS	1		/* Number of different flag states */

typedef struct flags_struct {
	int	flag[FLAGS];	/* Flags showing which inst present */
	int isSyscall[NPROC];
} flags_t;

#define NSYSARGS_host 20

typedef struct sysargs_struct {
	word_t sysarg[NPROC][NSYSARGS_host];
} arg_t;

typedef struct pe_struct {
	word_t	regs[REGSUSED];	/* Copy of regs for this PE */
	word_t	a;		/* Copy of accumulator */
	int	pc;		/* PC offset */
	int	ir;		/* IR */
} pe_t;

typedef struct data_struct {
	flags_t	flags;		/* Emulator flags */
	pe_t	pe[NPROC];	/* PE state info (fixed size) */
				/* Memory pool (potentially variable size) */
	word_t	mem[WARPDIV(NPROC)][DATASIZE][WARPSIZE];
} data_t;

typedef	struct {
	word_t	regs[WARPDIV(BNPROC)][REGSUSED][WARPSIZE];
} my_shared_t;

texture<unsigned short, 1, hipReadModeElementType> codetex;
texture<int, 1, hipReadModeElementType> cpooltex;

/*	Emulator as a single fragment function
*/
template <unsigned int blockSize>
__global__ void
emulate(register data_t *alldata, arg_t *hostsysargs)
{
	extern __shared__ my_shared_t	myshared[];
#define	BIPROC			(threadIdx.x)
	register int		warpmodbiproc = WARPMOD(BIPROC);
	register int		warpdivbiproc = WARPDIV(BIPROC);
	register const int	IPROC = BIPROC + (blockIdx.x * blockSize);
	register unsigned int	ir, op;
	register int		pc = alldata->pe[IPROC].pc;
	register word_t		a;
	register int		serial = 0;
	register volatile word_t	*mem = &(alldata->mem[WARPDIV(IPROC)][0][WARPMOD(BIPROC)]);
	register word_t		*regs = ((word_t *) &((*myshared).regs[warpdivbiproc][0][warpmodbiproc]));
	register int		moretodo;

#define	REGF(N)		regs[WARPMUL(REGNAME(N))].f
#define	REGI(N)		regs[WARPMUL(REGNAME(N))].i
#define	REGU(N)		regs[WARPMUL(REGNAME(N))].u

#define	MEMI(M)		mem[MAR(M)].i
#define	MEMH(M)		mem[MAR(M)].h[((M)&2)>>1]
#define	MEMB(M)		mem[MAR(M)].b[(M)&3]

	/*convert argBuf to GPU memory structure*/
	for(a.i=0; a.i<(sizeof(word_t)*MEMI(MOGSYM_NSYSARGS)); a.i=a.i+sizeof(word_t))
	 	MEMI(MOGSYM_SYSARGS+a.i) = hostsysargs->sysarg[IPROC][a.i/sizeof(word_t)].i;

	/* Reset flags */
	/*only one for now*/
	if (IPROC < FLAGS) alldata->flags.flag[IPROC] = 0;

	/* Restore registers */
	for (a.i=0; a.i<REGSUSED; ++a.i) {
		REGI(a.i) = alldata->pe[IPROC].regs[a.i].i;
	}
	a = alldata->pe[IPROC].a;

#define	CPOOL(p, i)	tex1D(cpooltex, CHASH(p-1, i))

#define	CODE(addr)	tex1D(codetex, addr)

//define	NEXT		{ op = OPCODE(ir = CODE(pc++)); }
#define	NEXT		{ ir = CODE(pc); op = OPCODE(ir); ++pc; }

#define	OPIS(o)		if (op == (OP##o)) { DO##o }

	/* Fetch current instruction */
	NEXT;

	do {
		/* Decode and execute single-instruction interpreters */
		moretodo = (TIMEOUT + 1);

		/* Expand-out the optimized interpreter sequence */
		OPORDER

		++serial;
		if (op != OPsys) moretodo = serial;
	} while (moretodo < TIMEOUT);

	__syncthreads();

	/* Is everybody stuck at a syscall? */
	if (op != OPsys) {
		/* Nope. */
		//alldata->flags.flag[0] = 0;
		alldata->flags.isSyscall[IPROC] = 0;
	}
	else { /*convert sysargs back to host format*/
		hostsysargs->sysarg[IPROC][0].i = MEMI(MOGSYM_NSYSARGS);
		for(a.i=0; a.i<(sizeof(word_t)*(MEMI(MOGSYM_NSYSARGS))); a.i=a.i+sizeof(word_t))
        	memcpy((void *)&hostsysargs->sysarg[IPROC][(a.i/sizeof(word_t))+1], (void *)&mem[MAR(MOGSYM_SYSARGS+a.i)],4);
		++pc;
		//alldata->flags.flag[0] = 1;
		alldata->flags.isSyscall[IPROC] = 1;
	}

	__syncthreads();

	/* Save registers */
	alldata->pe[IPROC].a = a;
	alldata->pe[IPROC].pc = pc - 1;
	for (a.i=0; a.i<REGSUSED; ++a.i) {
		alldata->pe[IPROC].regs[a.i].i = REGI(a.i);
	}
}

/*	Trace emulator as a single fragment function
*/
template <unsigned int blockSize>
__global__ void
emutrace(register data_t *alldata)
{
	extern __shared__ my_shared_t	myshared[];
#define	BIPROC			(threadIdx.x)
	register int		warpmodbiproc = WARPMOD(BIPROC);
	register int		warpdivbiproc = WARPDIV(BIPROC);
	register const int	IPROC = BIPROC + (blockIdx.x * blockSize);
	register unsigned int	ir, op;
	register int		pc = alldata->pe[IPROC].pc;
	register word_t	a;
	register int		serial = 0;
	register volatile word_t	*mem = &(alldata->mem[WARPDIV(IPROC)][0][WARPMOD(BIPROC)]);
	register word_t	*regs = ((word_t *) &((*myshared).regs[warpdivbiproc][0][warpmodbiproc]));
	register int		moretodo;

#define	REGF(N)		regs[WARPMUL(REGNAME(N))].f
#define	REGI(N)		regs[WARPMUL(REGNAME(N))].i
#define	REGU(N)		regs[WARPMUL(REGNAME(N))].u

#define	MEMI(M)		mem[MAR(M)].i
#define	MEMH(M)		mem[MAR(M)].h[((M)&2)>>1]
#define	MEMB(M)		mem[MAR(M)].b[(M)&3]


	/* Reset flags */
	if (IPROC < FLAGS) alldata->flags.flag[IPROC] = 0;

	/* Restore registers */
	for (a.i=0; a.i<REGSUSED; ++a.i) {
//	for (a.i=REGSUSED-1; a.i>=0; --a.i) {
		REGI(a.i) = alldata->pe[IPROC].regs[a.i].i;
	}
	a = alldata->pe[IPROC].a;


#define	CPOOL(p, i)	tex1D(cpooltex, CHASH(p-1, i))

#define	CODE(addr)	tex1D(codetex, addr)

//define	NEXT		{ op = OPCODE(ir = CODE(pc++)); }
#define	NEXT		{ ir = CODE(pc); op = OPCODE(ir); ++pc; }

#define	OPIS(o)		if (op == (OP##o)) { DO##o }

	/* Fetch current instruction */
	NEXT;

	do {
		/* Decode and execute single-instruction interpreters */
		moretodo = (TIMEOUT + 1);

#ifdef	TRACE
#define	TRACEELSE	else
#else
#define	TRACEELSE	/* don't serialize */
#endif

		OPIS(add)
TRACEELSE	OPIS(addf)
TRACEELSE	OPIS(and)
TRACEELSE	OPIS(div)
TRACEELSE	OPIS(divu)
TRACEELSE	OPIS(divf)
TRACEELSE	OPIS(xor)
TRACEELSE	OPIS(mul)
TRACEELSE	OPIS(mulf)
TRACEELSE	OPIS(or)
TRACEELSE	OPIS(slt)
TRACEELSE	OPIS(sltu)
TRACEELSE	OPIS(sltf)
TRACEELSE	OPIS(neg)
TRACEELSE	OPIS(negf)
TRACEELSE	OPIS(rem)
TRACEELSE	OPIS(remu)
TRACEELSE	OPIS(sra)
TRACEELSE	OPIS(sll)
TRACEELSE	OPIS(srl)

TRACEELSE	OPIS(j)
TRACEELSE	OPIS(jf)
TRACEELSE	OPIS(jt)

TRACEELSE	OPIS(i2f)
TRACEELSE	OPIS(u2f)
TRACEELSE	OPIS(f2i)

TRACEELSE	OPIS(li)
TRACEELSE	OPIS(lr)
TRACEELSE	OPIS(sr)
TRACEELSE	OPIS(lw)
TRACEELSE	OPIS(sw)
TRACEELSE	OPIS(lh)
TRACEELSE	OPIS(sh)
TRACEELSE	OPIS(lb)
TRACEELSE	OPIS(sb)

		++serial;
		if (op != OPsys) moretodo = serial;
	} while (moretodo < TIMEOUT);

	__syncthreads();

	/* Is everybody stuck at a syscall? */
	if (op != OPsys) {
		/* Nope. */
		alldata->flags.flag[0] = 1;
	}
	__syncthreads();

	/* Save registers */
	alldata->pe[IPROC].a = a;
	alldata->pe[IPROC].pc = pc - 1;
	for (a.i=0; a.i<REGSUSED; ++a.i) {
		alldata->pe[IPROC].regs[a.i].i = REGI(a.i);
	}
}

data_t	alldata;
arg_t hostsysargs;

/*	Status output for PE IPROC
*/
void
status(register const int IPROC)
{
	register int pc = alldata.pe[IPROC].pc;
	register int a = alldata.pe[IPROC].a.i;
	register int ir = textseg[pc];
	register int op = OPCODE(ir);
	register int i;

#undef	CPOOL
#define	CPOOL(PC,IR)	cpool[CHASH((PC), (IR))]
#define	IRIS(o)		{ if (op == (OP##o)) PR##o; }

	/* Dump registers */
	printf("PE%d: ", IPROC);
	printf("pc=0x%x", pc);
	printf(" ir=%c", '"');

		IRIS(sys)

		IRIS(add)
		IRIS(addf)
		IRIS(and)
		IRIS(div)
		IRIS(divu)
		IRIS(divf)
		IRIS(xor)
		IRIS(mul)
		IRIS(mulf)
		IRIS(or)
		IRIS(slt)
		IRIS(sltu)
		IRIS(sltf)
		IRIS(neg)
		IRIS(negf)
		IRIS(rem)
		IRIS(remu)
		IRIS(sra)
		IRIS(sll)
		IRIS(srl)

		IRIS(j)
		IRIS(jf)
		IRIS(jt)

		IRIS(i2f)
		IRIS(u2f)
		IRIS(f2i)

		IRIS(li)
		IRIS(lr)
		IRIS(sr)
		IRIS(lw)
		IRIS(sw)
		IRIS(lh)
		IRIS(sh)
		IRIS(lb)
		IRIS(sb)

	printf("%c a=0x%x", '"', a);
	for (i=0; i<REGSUSED; ++i) {
		printf(" $%d=%d",
		       i,
		       alldata.pe[IPROC].regs[i].i);
	}
	printf("\n");
}

int alldone(int* done)
{
	int total = 0;
	for(int i=0; i<NPROC; ++i)
		total += done[i];

	//printf("total: %i\n",total);
	if(total==NPROC)
		return 1;
	else 
		return 0;
}

int
main(int argc, char **argv) 
{
	register int i, j;
	int done[NPROC] = {0}; //used to determine exit condition
	data_t *gpudata;
	arg_t *gpusysargs;
	dim3 dimBlock(BNPROC, 1, 1);
	dim3 dimGrid((NPROC/BNPROC), 1, 1);

	/* Variables for a cuda timer */
	StopWatchInterface *timer = NULL;
	float exeTime = 0.0f;

	/* Create and reset a timer to measure the speed */
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);

	/* Check version and target */
	if (VERSION != version) {
		fprintf(stderr, "Error: code was compiled for version %d, not %d\n", version, VERSION);
		exit(1);
	}
	if (strcmp(TARGET, target) != 0) {
		fprintf(stderr, "Error: code was compiled for target %s, not %s\n", target, TARGET);
		exit(1);
	}

	/* Initialize everything... */
	for (i=0; i<NPROC; ++i) {
		/* Fill data memory, setting NPROC and IPROC */
		alldata.mem[WARPDIV(i)][0][WARPMOD(i)].i = NPROC;
		alldata.mem[WARPDIV(i)][1][WARPMOD(i)].i = i;
		for (j=2; j<DATASIZE; ++j) {
			alldata.mem[WARPDIV(i)][j][WARPMOD(i)].i = dataseg[j];
		}

		/* Initialize sp, ra, and other registers */
		for (j=2; j<REGSUSED; ++j) {
			alldata.pe[i].regs[j].i = 42;
		}
		alldata.pe[i].regs[0].i = DATASIZE-1;

		/* Set-up PE status regs */
		alldata.pe[i].pc = MOGSYM_main;
		alldata.pe[i].a.i = 0;

		/*initialize argbuf to nul(-1)*/
		for(j=0; j<NSYSARGS_host; j++)
			hostsysargs.sysarg[i][j].i=-1;
	}

		/* Allocate and copy alldata to GPU memory */
		checkCudaErrors( hipMalloc((void**) &gpudata, sizeof(data_t)) );
		checkCudaErrors( hipMemcpy(gpudata, &(alldata), sizeof(data_t), hipMemcpyHostToDevice) );

		/* Allocate and copy hostsysargs to GPU memory */
		checkCudaErrors( hipMalloc((void**) &gpusysargs, sizeof(arg_t)) );
		checkCudaErrors( hipMemcpy(gpusysargs, &(hostsysargs), sizeof(arg_t), hipMemcpyHostToDevice) );

	/* Allocate and copy cpool[] to GPU texture */
	hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<int>();
	hipArray* gputext2;
	checkCudaErrors(hipMallocArray( &gputext2, &channelDesc2, sizeof(cpool)/sizeof(int), 1));
	checkCudaErrors(hipMemcpyToArray( gputext2, 0, 0, cpool, sizeof(cpool), hipMemcpyHostToDevice));

	cpooltex.filterMode = hipFilterModePoint;
	cpooltex.normalized = false;    // access with unnormalized texture coordinates
	cpooltex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

	checkCudaErrors( hipBindTextureToArray( cpooltex, gputext2, channelDesc2));


	/* Allocate and copy textseg[] to GPU texture */
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned short>();
	hipArray* gputext;
	checkCudaErrors(hipMallocArray( &gputext, &channelDesc, sizeof(textseg)/sizeof(unsigned short), 1));
	checkCudaErrors(hipMemcpyToArray( gputext, 0, 0, textseg, sizeof(textseg), hipMemcpyHostToDevice));

	codetex.filterMode = hipFilterModePoint;
	codetex.normalized = false;    // access with unnormalized texture coordinates
	codetex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

	checkCudaErrors( hipBindTextureToArray( codetex, gputext, channelDesc));

	/* Time the main program execution */
	sdkStartTimer(&timer);

#ifdef	TRACE
	printf("flag = %d\n", alldata.flags.flag[0]);
	for (i=0; i<NPROC; ++i) {
		status(i);
	}
	do {
		/* Run the emulator for one instruction */
		emutrace<BNPROC><<< dimGrid, dimBlock, sizeof(my_shared_t) >>>(gpudata);

		/* Copy stuff back to host...
		   If we are doing a debug trace, copy entire data,
		   else could copy only the flags
		*/
		checkCudaErrors( hipMemcpy(&(alldata), gpudata, sizeof(data_t), hipMemcpyDeviceToHost) );
		printf("flag = %d\n", alldata.flags.flag[0]);
		for (i=0; i<NPROC; ++i) {
			status(i);
		}
	} while (alldata.flags.flag[0] != 0);
#else
	do {
		/* Run the emulator for a while */
		emulate<BNPROC><<< dimGrid, dimBlock, sizeof(my_shared_t) >>>(gpudata, gpusysargs);

		/* Copy stuff back to host...
		   If we are doing a debug trace, copy entire data,
		   else could copy only the flags
		*/
		checkCudaErrors( hipMemcpy(&(alldata), gpudata, sizeof(flags_t), hipMemcpyDeviceToHost) );

		//test
		checkCudaErrors( hipMemcpy(&(hostsysargs), gpusysargs, sizeof(arg_t), hipMemcpyDeviceToHost) );

		//if(alldata.flags.flag[0] == 1) {/*if there was a syscall start decoding*/
		/*copy sysargs back to host*/
		for(i=0; i<NPROC; i++)	{
			int nsysargs = hostsysargs.sysarg[i][0].i;
			int sysCallNum = hostsysargs.sysarg[i][1].i;
			/*look in argument buffers, decode system call, execute it within some environment, then return data*/
			if(alldata.flags.isSyscall[i] == 1) {
				switch(sysCallNum) {
					case 0: {/*exit()*/
						done[i]=1;
					}
					break;
					case 1:	{/*time(time_t *t)*/
						hostsysargs.sysarg[i][0].i = time(NULL);
						printf("time: %i\n", hostsysargs.sysarg[i][2].i);
					}
					break;
					case 2: {/*dup(int filedes)*/
						int filedes=hostsysargs.sysarg[i][2].i;
	                    printf("filedes: %i\n",filedes);
						hostsysargs.sysarg[i][0].i = filedes;
					}
					break;
					case 3: {/*putchar(char c)*/
						char c=(char)hostsysargs.sysarg[i][2].i;
						putchar(c);
					}
					break;
					case 4: {/*getchar()*/
						hostsysargs.sysarg[i][0].i = getchar();
					}
					break;
					case 5: {/*open(const char *pathname, int flags)*/
						char *pathname = (char *)malloc((nsysargs-3)*sizeof(char));
						int ichar;
						for(ichar=0; ichar<(nsysargs-3); ++ichar)
							pathname[ichar] = (char)hostsysargs.sysarg[i][ichar+2].i;
						int fileflags = hostsysargs.sysarg[i][ichar+2].i;
						int filemode = hostsysargs.sysarg[i][ichar+3].i;
						printf("pathname: %s\nfileflags: %i\nfilemode: %i\n",pathname,fileflags,filemode);
						int ro = open(pathname,fileflags,filemode);
						hostsysargs.sysarg[i][0].i = ro;
						printf("openfd: %i\n",ro);
					}
					break;
					case 6: {/*close(int fd)*/
						int fd = hostsysargs.sysarg[i][2].i;
						close(fd);
					}
					break;
					case 7: {/*int read(int fd, void *buf, int count)*/
						int fd = hostsysargs.sysarg[i][2].i;
						int count = hostsysargs.sysarg[i][3].i;
						read(fd,(void *)(&hostsysargs.sysarg[i][0]),count);
						printf("read: count: %i fd: %i buf: %s\n",count,fd,(char *)(&hostsysargs.sysarg[i][0]));
            
					}
					break;
					case 8: {/*int write(int fd, const void *buf, int count)*/ 
						int fd = hostsysargs.sysarg[i][2].i;
						int count = hostsysargs.sysarg[i][3].i;
						char buf[count];
						memcpy((void *)buf,(void *)(&hostsysargs.sysarg[i][4]),count);
						printf("write: count: %i fd: %i buf: %s other: %i %i %i\n",count,fd,buf,buf[0],buf[1],buf[2]);
						write(fd,buf,count);
					}
					break;
					case 9: {/*putint(int i)*/
						int anint=hostsysargs.sysarg[i][2].i;
						printf("%i\n",anint);
					}
					break;
					default:
					break;
				}
			}
		}
		checkCudaErrors( hipMemcpy(gpusysargs, &(hostsysargs), sizeof(arg_t), hipMemcpyHostToDevice) );
	} while (!alldone(done));
#endif

	checkCudaErrors( hipMemcpy(&(alldata), gpudata, sizeof(data_t), hipMemcpyDeviceToHost) );

	/* Stop the timer */
	sdkStopTimer(&timer);

	/* Print the execution time of the mulate function (and it's loop/readback) */
	exeTime = sdkGetTimerValue(&timer);
	printf("\nTotal emulate time (without setup): %.3f s\n",exeTime/1000.0f);

        checkCudaErrors(hipFree(gpudata));
	sdkDeleteTimer(&timer);
}
